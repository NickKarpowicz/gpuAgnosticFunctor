#include <iostream>
#include <chrono>
#include <vector>
#ifdef __HIPCC__
    #include "cudaDeviceHeader.cuh"
    typedef CUDAdevice device;
//#elif defined USESYCL
#else
    #include "syclDeviceHeader.hpp"
    typedef SYCLdevice device;    

    std::cout << "Scanning for devices...\n";
    for (const auto& p : sycl::platform::get_platforms()) {
        for (const auto& d : p.get_devices()) {
            std::cout << "Found: " << d.get_info<sycl::info::device::name>() << '\n';
        }
    }
// #else
//     #include "ompDeviceHeader.hpp"
//     typedef OMPdevice device;
#endif

class setValues{
public:
    float* A; 
    float* B;
    deviceFunction void operator()(const unsigned int i) const {
        A[i] = 0.1f * static_cast<float>(i);
        B[i] = 2.1f * A[i] + 1.0f;
    }
};

class multiplyAbyB{
public:
    float* A;
    float* B;
    deviceFunction void operator()(const unsigned int i) const{
        A[i] *= B[i];
    }
};

class divideAbyB{
public:
    float* A; 
    float* B;
    deviceFunction void operator()(const unsigned int i) const{
        A[i] /= B[i];
    }
};

int main(){
    unsigned int vectorSize = 8192*65536;
    unsigned int repetitions = 32;
    unsigned int Nthreads = 64;
    unsigned int Nblocks = vectorSize/Nthreads;
    float* deviceA;
    float* deviceB;
    std::vector<float> cpuA(vectorSize, 0.0f);

    auto timerBegin = std::chrono::high_resolution_clock::now();
    device d;
    d.Malloc((void**)&deviceA, vectorSize * sizeof(float));
    d.Malloc((void**)&deviceB, vectorSize * sizeof(float));
    d.LaunchKernel(Nblocks, Nthreads, setValues{deviceA, deviceB});
    for(unsigned int i = 0; i < repetitions; ++i){
        d.LaunchKernel(Nblocks, Nthreads, multiplyAbyB{deviceA, deviceB});
        d.LaunchKernel(Nblocks, Nthreads, divideAbyB{deviceA, deviceB});
    }
    d.MemcpyDeviceToHost(cpuA.data(),deviceA,vectorSize*sizeof(float));
    d.Free(deviceA);
    d.Free(deviceB);

    auto timerEnd = std::chrono::high_resolution_clock::now();
    std::cout << 
    "Took " << 
    1e-3 * static_cast<double>(
        std::chrono::duration_cast<std::chrono::microseconds>
        (timerEnd - timerBegin).count()) << 
        " ms\n";

    for(int i = 0; i<10; ++i){
        std::cout << i << ": " << cpuA[i] <<"\n"; 
    }
}