#include <iostream>
#include <chrono>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void setValues(
    float* A, 
    float* B)
    {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    A[i] = 0.1f * static_cast<float>(i);
    B[i] = 2.1f * A[i] + 1.0f;
}

__global__ void multiplyAbyB(
    float* A, 
    float* B)
    {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    A[i] *= B[i];
}

__global__ void divideAbyB(
    float* A, 
    float* B)
    {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    A[i] /= B[i];
}

int main()
{
    unsigned int vectorSize = 8192*65536;
    unsigned int repetitions = 32;
    unsigned int Nthreads = 64;
    unsigned int Nblocks = vectorSize/Nthreads;
    float* deviceA;
    float* deviceB;
    std::vector<float> cpuA(vectorSize, 0.0f);

    auto timerBegin = std::chrono::high_resolution_clock::now();
    
    hipMalloc(&deviceA, vectorSize * sizeof(float));
    hipMalloc(&deviceB, vectorSize * sizeof(float));
    setValues<<<Nblocks, Nthreads>>>(deviceA, deviceB);
    for(int i = 0; i < repetitions; ++i){
        multiplyAbyB<<<Nblocks, Nthreads>>>(deviceA, deviceB);
        divideAbyB<<<Nblocks, Nthreads>>>(deviceA, deviceB);
    }
    hipDeviceSynchronize();
    hipMemcpy(cpuA.data(),deviceA,vectorSize*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(deviceA);
    hipFree(deviceB);
    
    auto timerEnd = std::chrono::high_resolution_clock::now();
    
    std::cout << 
    "Took " << 
    1e-3 * static_cast<double>(
        std::chrono::duration_cast<std::chrono::microseconds>
        (timerEnd - timerBegin).count()) << 
    " ms\n";
    
    for(int i = 0; i<3; ++i){
        std::cout << i << ": " << cpuA[i] <<"\n"; 
    }
}