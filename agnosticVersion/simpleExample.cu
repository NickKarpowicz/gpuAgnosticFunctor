#include <iostream>
#include <chrono>
#include <vector>
#ifdef __HIPCC__
    #include "cudaDeviceHeader.cuh"
    typedef CUDAdevice device;
#elif defined USESYCL
    #include "syclDeviceHeader.hpp"
    typedef SYCLdevice device;    
#else
    #include "ompDeviceHeader.hpp"
    typedef OMPdevice device;
#endif

class setValues{
public:
    float* A; 
    float* B;
    deviceFunction void operator()(const unsigned int i) const {
        A[i] = 0.1f * static_cast<float>(i);
        B[i] = 2.1f * A[i] + 1.0f;
    }
};

class multiplyAbyB{
public:
    float* A;
    float* B;
    deviceFunction void operator()(const unsigned int i) const{
        A[i] *= B[i];
    }
};

class divideAbyB{
public:
    float* A; 
    float* B;
    deviceFunction void operator()(const unsigned int i) const{
        A[i] /= B[i];
    }
};

int main(){
    unsigned int vectorSize = 65536;
    unsigned int repetitions = 65536;
    unsigned int Nthreads = 64;
    unsigned int Nblocks = vectorSize/Nthreads;
    float* deviceA;
    float* deviceB;
    std::vector<float> cpuA(vectorSize, 0.0f);

    auto timerBegin = std::chrono::high_resolution_clock::now();
    device d;
    d.Malloc((void**)&deviceA, vectorSize * sizeof(float));
    d.Malloc((void**)&deviceB, vectorSize * sizeof(float));
    d.LaunchKernel(Nblocks, Nthreads, setValues{deviceA, deviceB});
    for(int i = 0; i< repetitions; ++i){
        d.LaunchKernel(Nblocks, Nthreads, multiplyAbyB{deviceA, deviceB});
        d.LaunchKernel(Nblocks, Nthreads, divideAbyB{deviceA, deviceB});
    }
    d.MemcpyDeviceToHost(cpuA.data(),deviceA,vectorSize*sizeof(float));
    d.Free(deviceA);
    d.Free(deviceB);

    auto timerEnd = std::chrono::high_resolution_clock::now();
    std::cout << 
    "Took " << 
    1e-3 * static_cast<double>(
        std::chrono::duration_cast<std::chrono::microseconds>
        (timerEnd - timerBegin).count()) << 
        " ms\n";

    for(int i = 0; i<10; ++i){
        std::cout << i << ": " << cpuA[i] <<"\n"; 
    }
}